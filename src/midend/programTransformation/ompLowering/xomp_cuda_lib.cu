#include "hip/hip_runtime.h"
/* A simple lib to facilitate CUDA coding 

Extensions to the xomp interface

Liao 4/11/2012
*/

//#include "cuda_lib.h"
#include "libxomp.h"
//----------------------------------------------------
// Device xomp_cuda_property retrieving functions

extern struct DDE_data * DDE_head;
extern struct DDE_data * DDE_tail;

hipDeviceProp_t* xomp_cuda_prop = NULL; 
bool xomp_verbose = false;

// this can be called multiple times. But the xomp_cuda_prop variable will only be set once
hipDeviceProp_t * xomp_getCudaDeviceProp()
{
  if (xomp_cuda_prop == NULL )
  {
    xomp_cuda_prop = (hipDeviceProp_t *) malloc(sizeof(hipDeviceProp_t));
    assert (xomp_cuda_prop != NULL);
    int count;
    hipGetDeviceCount (&count);
    assert (count>=1); // must have at least one GPU here
    hipGetDeviceProperties  (xomp_cuda_prop, 0);
  }
  return xomp_cuda_prop;
}

void xomp_print_gpu_info()
{
  int max_threads_per_block = xomp_getCudaDeviceProp()->maxThreadsPerBlock;
  int max_blocks_per_grid_x = xomp_getCudaDeviceProp()->maxGridSize[0];
  int global_memory_size =    xomp_getCudaDeviceProp()->totalGlobalMem;
  int shared_memory_size =    xomp_getCudaDeviceProp()->sharedMemPerBlock;
  int registers_per_block =   xomp_getCudaDeviceProp()->regsPerBlock;

  printf ("Found a GPU with \n\tmax threads per block=%d, \n\tmax blocks for Grid X dimension=%d\n\
      \tglobal mem bytes =%d, \n\tshared mem bytes =%d, \n\tregs per block = %d\n",
      max_threads_per_block, max_blocks_per_grid_x, global_memory_size ,  shared_memory_size,
      registers_per_block);
}
// A helper function to probe physical limits based on GPU Compute Capability numbers
// Reference: http://developer.download.nvidia.com/compute/cuda/CUDA_Occupancy_calculator.xls
size_t xomp_get_maxThreadBlocksPerMultiprocessor()
{
  int major, minor; 
  major = xomp_getCudaDeviceProp()-> major;
  minor = xomp_getCudaDeviceProp()-> minor;
  if (major <= 2) //1.x and 2.x: 8 blocks per multiprocessor
    return 8;
  else if (minor <= 5)
    return 16;
  else
  {
   printf("Error: xomp_get_maxThreadBlocksPerMultiprocessor(): unhandled Compute Capability numbers%d.%d \n", major, minor);
   assert (false);
  }
  assert (false);
  return 0;
}

// max thread per block, useful for 1-D problem
// The goal is to maximize GPU occupancy for each multiprocessor : physical max warps 
// Reference: http://developer.download.nvidia.com/compute/cuda/CUDA_Occupancy_calculator.xls
//
// Two physical limits are considered for now
//  1) max-active-threads per multiprocessor 
//  2) max active thread blocks per multiprocessor
// So for 1-D block, max threads per block = maxThreadsPerMultiProcessor /  maxBlocks per multiprocessor
size_t xomp_get_maxThreadsPerBlock()
{
  // this often causes oversubscription to the cores supported by GPU SM processors
  //return xomp_getCudaDeviceProp()->maxThreadsPerBlock;
  //return 128;
  // 2.0: 1536/8= 192 threads per block
  // 3.5 2048/16 = 128
  return xomp_getCudaDeviceProp()->maxThreadsPerMultiProcessor / xomp_get_maxThreadBlocksPerMultiprocessor();
}

/*
* In order to ensure best performance, we setup max_block limitation here, so that each core in the GPU works on only one threads.
* Use XOMP_accelerator_loop_default() runtime to support input data size that exceeds max_block*xomp_get_maxThreadsPerBlock().  
*/
size_t xomp_get_max1DBlock(size_t s)
{
#if 1  
  size_t block_num = s/xomp_get_maxThreadsPerBlock();
  if (s % xomp_get_maxThreadsPerBlock()!= 0)
     block_num ++;
  //return block_num;     

  size_t max_block = xomp_getCudaDeviceProp()->multiProcessorCount* xomp_get_maxThreadBlocksPerMultiprocessor();

  return block_num<max_block? block_num: max_block; 

  /* max threads per multiprocessor / threads-per-block  * num_multiprocessor */
  //return xomp_getCudaDeviceProp()->multiProcessorCount*(xomp_getCudaDeviceProp()->maxThreadsPerMultiProcessor /xomp_get_maxThreadsPerBlock()) ;
  //return xomp_getCudaDeviceProp()->maxThreadsPerMultiProcessor /xomp_get_maxThreadsPerBlock() ;
#else
  return xomp_getCudaDeviceProp()->multiProcessorCount* xomp_get_maxThreadBlocksPerMultiprocessor();
#endif
}

// Get the max number threads for one dimension (x or y) of a 2D block
// Two factors are considered: the total number of threads within the 2D block must<= total threads per block
//  x * y <= maxThreadsPerBlock 512 or 1024
// each dimension: the number of threads must <= maximum x/y-dimension
//    x <= maxThreadsDim[0],  1024
//    y <= maxThreadsDim[1], 1024 
//  maxThreadsDim[0] happens to be equal to  maxThreadsDim[1] so we use a single function to calculate max segments for both dimensions
size_t xomp_get_max_threads_per_dimesion_2D ()
{

  int max_threads_per_block = xomp_getCudaDeviceProp()->maxThreadsPerBlock;
  // we equalize the number of threads in each dimension
  int max_threads_per_2d_dimension = (int)(sqrt((float)max_threads_per_block));  
  assert (max_threads_per_2d_dimension*max_threads_per_2d_dimension<= max_threads_per_block);

  // our assumption is that dim[0] == dim[1] so we handle x and y in one function
  assert ( xomp_getCudaDeviceProp()->maxThreadsDim[0] == xomp_getCudaDeviceProp()->maxThreadsDim[1]);   
  assert (max_threads_per_2d_dimension <= xomp_getCudaDeviceProp()->maxThreadsDim[0]);
  return max_threads_per_2d_dimension;
}

// return the max number of segments for a dimension (either x or y) of a 2D block
// we define the number of segments to be  SIZE_of_Dimension_x/max_threads_x_dimension
size_t xomp_get_maxSegmentsPerDimensionOf2DBlock(size_t dimension_size)
{
  // For simplicity, we don't yet consider the factor of warp size for now
  // TODO: block size should be divisible by the warp size??
  // e.g. max threads per block is 1024, then max number of tiles per dimension in a 2D block is 1024^0.5 = 32 threads
  size_t max_threads_per_2d_dimension = xomp_get_max_threads_per_dimesion_2D ();
  size_t block_num_x_or_y =  dimension_size/max_threads_per_2d_dimension;
  if (dimension_size % max_threads_per_2d_dimension != 0)
     block_num_x_or_y ++;

  return block_num_x_or_y;
}

/*-----------------------------------------------------
   Device memory allocation functions 
*/
void* xomp_deviceMalloc(size_t size)
{
  void * devPtr;
  hipError_t rt =  hipMalloc(&devPtr, size);
  if ( (size !=0) && (rt == hipSuccess) )
  {
    return devPtr;
  }
  else
  {
    fprintf(stderr, "Error: hipMalloc() failed to allocate the requested %d bytes!\n",size );
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

// A host version
void* xomp_hostMalloc(size_t size)
{
  assert (size>0);
  void* hostPtr;
  hostPtr = (char*) malloc (size);
  if (hostPtr == NULL)
  {
    fprintf(stderr, "Error: malloc() failed to allocate the requested %d bytes!\n",size );
    assert (hostPtr != NULL);
  }
  return hostPtr; 
}

// memory copy from src to dest, return the pointer to dest. NULL pointer if anything is wrong 
void * xomp_memcpyHostToDevice (void *dest, const void * src, size_t n)
{
  assert (dest != NULL);
  assert (src != NULL);
  if (xomp_verbose)
    printf("xomp_memcpyHostToDevice(): dest=%p src =%p size=%d\n",dest, src, n);
  hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyHostToDevice);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy memory from Host %p to Device %p, for %d bytes!\n",src, dest, n);
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

void * xomp_memcpyDeviceToHost (void *dest, const void * src, size_t n)
{
  assert (dest != NULL);
  assert (src != NULL);
  if (xomp_verbose)
    printf("xomp_memcpyDeviceToHost(): dest=%p src =%p size=%d\n",dest, src, n);
  hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyDeviceToHost);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy memory from Device %p to Host %p, for %d bytes!\n",src, dest, n);
    fprintf(stderr, "Error message is =%s\n",hipGetErrorString(rt));
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

// copy a dynamically allocated host source array to a linear dest address on a GPU device. 
// The dimension information of the source array is given by: int dimensions[dimension_size], with known element size. 
// bytes_copied reports the total bytes copied by this function.  
// Liao 4/25/2012
void * xomp_memcpyDynamicHostToDevice (void *dest, const void * src, int * dimensions, size_t dimension_size, size_t element_size, size_t *bytes_copied)
{
  assert (dest != NULL);
  assert (src != NULL);
  if  (dimension_size == 1)  // down to the final, inner-most dimension 
  {
    // this is a firm, bottom count of bytes copied
    *bytes_copied = element_size* dimensions[0]; // number of elements * element_size
    xomp_memcpyHostToDevice (dest, src, *bytes_copied);
  }
  else 
  { 
    assert (dimension_size>=2);
    // 2-D or more: arrays of sub-arrays, copy each sub-array separately
    void ** array2 = (void**) src ; // re-interpret src to be array of arrays
    size_t sub_array_count = dimensions[0];  // top dimension, how many sub-arrays

    // prepare dimension information for sub-arrays
    dimensions ++;// go to the next dimension
    dimension_size --; // sub-array has one less dimension
    char* new_dest = (char*) dest; // byte addressable for dest
    size_t total_subarray_bytes_copied =0; 
    int i; 
    for (i = 0; i< sub_array_count; i++) // for each sub-arrays
    {
      size_t subarray_bytes_copied = 0; 
      void* sub_array_src = (void*) (array2[i]);  // get start address for each sub-array
      assert (sub_array_src != NULL);
      // recursively call to copy each sub-array
      xomp_memcpyDynamicHostToDevice (new_dest, (void*)sub_array_src, dimensions, dimension_size, 
          element_size, &subarray_bytes_copied);

      total_subarray_bytes_copied += subarray_bytes_copied; // recursively accumulate the bytes copied, instead of calculating directly.
      new_dest += subarray_bytes_copied ; // update the dest offset here
    }
    *bytes_copied = total_subarray_bytes_copied; 
  }
  return dest; 
}


// copy linear src memory to dynamically allocated destination, with dimension information given by
// int dimensions[dimension_size]
// the source memory has total n continuous memory, with known size for each element
// the total bytes copied by this function is reported by bytes_copied
// test code: mallocArray-xomp.cu
void * xomp_memcpyDynamicDeviceToHost (void *dest, int * dimensions, size_t dimension_size, const void * src, size_t element_size, size_t *bytes_copied)
{
  // int dimensions[1] = {10}; dimension_size =1; 
  // a[10]: 
  if  (dimension_size == 1)  // down to the final dimension 
  {
    // this is a firm, bottom count of bytes copied
    *bytes_copied = element_size* dimensions[0]; // number of elements * element_size
    xomp_memcpyDeviceToHost (dest, src, *bytes_copied);
  }
  else 
  { 
    int i; 
    assert (dimension_size>=2);
    // 2-D or more: arrays of sub-arrays, copy each sub-array separately
    void ** array2 = (void**) dest; // re-interpret dest to be array of arrays
    size_t sub_array_count = dimensions[0];  // current dimension, how many sub-arrays

    // prepare dimension information for sub-arrays
    dimensions ++;// go to the next dimension
    dimension_size --;
    char* new_src = (char*) src; // byte addressable for src
    size_t total_subarray_bytes_copied =0; 
    for (i = 0; i< sub_array_count; i++) // for each sub-arrays
    {
      size_t subarray_bytes_copied =0; 
      void* sub_array_dest = (void*) (array2[i]);  // get start address for each sub-array

      xomp_memcpyDynamicDeviceToHost ((void*)sub_array_dest,  dimensions, dimension_size, 
          new_src, element_size, &subarray_bytes_copied);

      total_subarray_bytes_copied += subarray_bytes_copied; // recursively accumulate the bytes copied, instead of calculating directly.
      new_src += subarray_bytes_copied ; // update the source offset here
    }
    *bytes_copied = total_subarray_bytes_copied; 
  }
  return dest; 
}

void * xomp_memcpyDeviceToDevice (void *dest, const void * src, size_t n)
{
    hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyDeviceToDevice);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy from Device %p to Device %p for requested %d bytes!\n", src, dest, n );
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }
}

void * xomp_memcpyHostToHost (void *dest, const void * src, size_t n) // same as memcpy??
{
    hipError_t rt = hipMemcpy (dest, src, n, hipMemcpyHostToHost);    
  if (rt == hipSuccess)
    return dest;
  else
  {
    fprintf(stderr, "Error: hipMemcpy() failed to copy from Host %p to Host %p for requested %d bytes!\n", src, dest, n );
    assert (false);
    return NULL; // it is a bad idea to silently return a NULL pointer
  }

}

//------------------------------------------------------
// free the device memory pointed by a pointer, return false in case of failure, otherwise return true
bool xomp_freeDevice(void* devPtr)
{
  hipError_t rt = hipFree (devPtr);
  if (rt == hipSuccess)
    return true;
  else
    return false;
}

// free the host memory pointed by a pointer, return false in case of failure, otherwise return true
bool xomp_freeHost(void* hostPtr)
{
  hipError_t rt = hipHostFree(hostPtr);
  if (rt == hipSuccess)
    return true;
  else
    return false;
}

//------------------------------------------------------
// data set size checking functions
#if 0
  // make sure the length of the array can be mapped to the cuda threads
  assert (SIZE <= max_blocks_per_grid_x* max_threads_per_block);
  // make sure the data will fit into the device memory (shared memory)
  printf("matrix-vector multiplication with size=%d\n", SIZE);
  // one matrix and two vectors
  int mem_required = SIZE*SIZE*sizeof(float) + SIZE* sizeof(float) *2;
  if (global_memory_size > 0) //sometimes the number is too large and it overflows to be a negative integer
    assert (mem_required  <= global_memory_size);
#endif

//------------------------------------------------------
#if 0
double xomp_time_stamp()
{
  struct timeval t;
  double time;
  gettimeofday(&t, NULL);
  time = t.tv_sec + 1.0e-6*t.tv_usec;
  return time;
}
#endif
//------------------------------------------------------
// Host side helper functions
//--- a helper function to allocate 2-D arrays
/* Allocate a multi-dimensional array
 *
 * Input parameters:
 *  int *dimensions:  an integer array storing the size of each dimension
 *  size_t dimension_num: the number of dimensions
 *  size_t esize: the size of an array element
 *
 * return:
 *  the pointer to the allocated array
 * */
void * xomp_mallocArray(int * dimensions, size_t dimension_num, size_t esize)
{
  int i;
  void * array = NULL;

//  if (xomp_verbose)
//    printf("xomp_xomp_mallocArray(): dimensions=%p dimension =%d element size=%d\n",dimensions, dimension_num, esize);
  //  Handle 1-D array: do element-wise malloc
  if ( dimension_num == 1)
  {
    array = (void *) malloc(dimensions[dimension_num - 1]*esize);
    if(array == NULL)
    {
      fprintf(stderr, "out of memory\n");
      abort();
    }
  }
  else // two and more dimensions to be allocated: reduce it to be a 2-step allocation
  {
    // 1st step: allocate the first dimension 
    //           by treating it as allocating a 1-D array of arrays (pointer)
    void ** array2 = NULL;
    array2 = (void **) xomp_mallocArray(dimensions, 1 ,sizeof (void *));
    size_t prev_dim_size = dimensions[0];// number of elements of the first dimension

    // 2nd step: allocate the remaining N -1 dimension arrays, each is an element of the first array 
    // peel off the 1st(previous) dimension, focus on the rest dimensions
     dimensions ++;
    // each of element is an array has a smaller dimension number
     dimension_num --;
    for(i = 0; i < prev_dim_size ; i++)
    {
      array2[i] = xomp_mallocArray (dimensions, dimension_num, esize);
    }
    // return the pointer to the first dimension
    array = (void *) array2;
  }
  return array;
}


/* Free a pointer to a multi-dimensional array
 * int * dimensions: store the sizes of each dimension
 * size_t dimension_num: the number of dimensions
 *
 * */
void xomp_freeArrayPointer (void* array, int * dimensions, size_t dimension_num)
{

  int i;
  // 1-D case, call free() directly
  if (dimension_num == 1)
  {
    free (array);
  }
  else
  { // 2-D or more, iterate through higher dimension and try to free inner arrays
    int prev_dim_size = dimensions [0];
    // step into one dimension
    dimensions ++;
    dimension_num --;
    for (i =0; i< prev_dim_size ; i++)
    {
      xomp_freeArrayPointer (((void **)array)[i], dimensions, dimension_num);
    }
  }
}

#if 0
/*  reduction minus is handled the same way as reduction plus since we just replace the reduction variable with its local copy for each thread
    The associated statement is intact except for the variable replacement : e.g. a-=5 becomes local_a -= 5;
    in the end of each thread accumulates thread local negative values.
    At the block level, we just simply add them all to be the block level negative values
*/
/* we have to encode the type into function name since C function signature does not include parameter list! */
#define XOMP_INNER_BLOCK_REDUCTION_DEF(dtype) \
__device__ void xomp_inner_block_reduction_##dtype(dtype local_value, dtype * grid_level_results, int reduction_op) \
{ \
  /* __shared__ float* sdata[gridDim.x]; not compilable */ \
  /* block size of data, size is specified by the kernel launch parameter (3rd one) */ \
  /* shared data has to have different names for different types. Cannot reuse name across types. */ \
  extern __shared__ dtype sdata_##dtype[];  \
  sdata_##dtype[threadIdx.x] = local_value;  \
  __syncthreads(); \
  /* blockDim.x is the block size */ \
  int isEvenSize = (blockDim.x % 2 ==0); \
  /* contiguous range pattern: half folding and add */ \
  for(int offset = blockDim.x / 2; \
      offset > 0;    /* folding and add */ \
      offset >>= 1) /* offset shrinks half each time */ \
  { \
    if(threadIdx.x < offset)  \
    { \
      /* add a partial sum upstream to our own */ \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
            sdata_##dtype[threadIdx.x] += sdata_##dtype[threadIdx.x + offset]; \
            break; \
         /*  TODO add support for more operations*/ \
         default:  \
            { \
              /* TODO: add assertion or set hipError_t with an error code */ \
              /* cannot call a host function */ \
              /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
              /* assert (false); */ \
             } \
      } /* end switch */ \
    } \
    /* remember to handle the left element */ \
    if ((threadIdx.x == 0) && !isEvenSize) \
    { \
      switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          sdata_##dtype[0]+= sdata_##dtype[2*offset];  \
          break; \
        /* TODO add more operation support */  \
        default: \
          {  \
            /* TODO: add assertion or set hipError_t with an error code */  \
            /* cannot call a host function */ \
            /* fprintf (stderr, "Error. xomp_inner_block_reduction() unhandled reduction operation:%d\n",reduction_op); */ \
            /* assert (false); */ \
          } \
      } /* end switch */ \
    } \
    isEvenSize = ( offset % 2 ==0); /* prepare next round*/ \
    /* MUST wait until all threads in the block have updated their partial sums */ \
    __syncthreads(); /* sync after each folding */ \
  } \
  /* thread 0 writes the final result to the partial sum of this thread block */ \
  if(threadIdx.x == 0) \
  { \
    grid_level_results[blockIdx.x] = sdata_##dtype[0]; \
  } \
}

XOMP_INNER_BLOCK_REDUCTION_DEF(int)
XOMP_INNER_BLOCK_REDUCTION_DEF(float)
XOMP_INNER_BLOCK_REDUCTION_DEF(double)

#undef XOMP_INNER_BLOCK_REDUCTION_DEF 

#endif

// TODO: handle more different reduction operations
// TODO : add assertion support
#define XOMP_BEYOND_BLOCK_REDUCTION_DEF(dtype) \
  dtype xomp_beyond_block_reduction_##dtype(dtype * per_block_results, int numBlocks, int reduction_op) \
{ \
  dtype result ; \
  dtype* per_block_results_cpu = (dtype *)xomp_hostMalloc (numBlocks*sizeof(dtype)); \
  xomp_memcpyDeviceToHost (per_block_results_cpu, per_block_results, sizeof(dtype)* numBlocks); \
  int r_i; \
  for (r_i =1; r_i < numBlocks; r_i++) \
  { \
     switch (reduction_op){ \
        case XOMP_REDUCTION_PLUS: \
        case XOMP_REDUCTION_MINUS: \
          per_block_results_cpu[0]+= per_block_results_cpu[r_i]; \
          break; \
        default: \
          { \
          } \
      } \
  } \
  result = per_block_results_cpu[0]; \
  xomp_freeHost(per_block_results_cpu); \
  return result; \
} 

//TODO define more types of CPU level reduction support
XOMP_BEYOND_BLOCK_REDUCTION_DEF(int)
XOMP_BEYOND_BLOCK_REDUCTION_DEF(float)
XOMP_BEYOND_BLOCK_REDUCTION_DEF(double)

#undef XOMP_BEYOND_BLOCK_REDUCTION_DEF 

/* some of the ompacc runtime API */
int omp_get_num_devices() {
  int deviceCount = 0;
  hipGetDeviceCount(&deviceCount);
  return deviceCount;
}

//! A helper function to copy a mapped variable from src to desc
void copy_mapped_variable (struct XOMP_mapped_variable* desc, struct XOMP_mapped_variable* src)
{
  assert (src != NULL);
  assert (desc != NULL);

  desc->address = src->address; 
  desc->size= src->size; 
  desc->dev_address = src ->dev_address; 
   // we do not want to inherit the copy directions or map-type of parent DDE's variable
   // OpenMP 4.0 has the reuse enclosing data and discard map-type rule.
  //desc->copyFrom= src ->copyFrom; 
}

// create a new DDE-data node and 
// append it to the end of the tracking list, and 
// copy all variables from its parent node to be into the set of inherited variable set.
void xomp_deviceDataEnvironmentEnter()
{
  // create a new DDE node and initialize it
  struct DDE_data * data = (struct DDE_data *) malloc (sizeof (struct DDE_data));
  assert (data!=NULL);
  data->new_variable_count = 0;
  data->inherited_variable_count = 0;
  data->parent = NULL;
  data->child= NULL;

  // For simplicity, we pre-allocate the storage for the list of variables
  // TODO: improve the efficiency
  data->new_variables = (struct XOMP_mapped_variable*) malloc (XOMP_MAX_MAPPED_VARS * sizeof (struct XOMP_mapped_variable));
  data->inherited_variables = (struct XOMP_mapped_variable*) malloc (XOMP_MAX_MAPPED_VARS * sizeof (struct XOMP_mapped_variable));

  // Append the data to the list
  // Case 1: empty list, add as the first node, nothing else to do
  if (DDE_tail == NULL)
  {
    assert (DDE_head == NULL );
    DDE_head = data;
    DDE_tail = data;
    return; 
  }

  // Case 2: non-empty list
  // create double links
  data->parent = DDE_tail; 
  DDE_tail->child = data;
  // shift the tail
  DDE_tail = data;

  // copy all variables from its parent node into the inherited variable set. 
  // Both new and inherited variables of the parent node become inherited for the current node
  data->inherited_variable_count = data->parent->new_variable_count + data->parent->inherited_variable_count;
  data->inherited_variables = (struct XOMP_mapped_variable*) malloc (data->inherited_variable_count * sizeof (struct XOMP_mapped_variable));
  assert (data->inherited_variables != NULL);

  int i;
  int offset = 0;
  for (i = 0; i < data->parent->new_variable_count; i++)
  {
    struct XOMP_mapped_variable* dest_element  = data->inherited_variables + offset;
    struct DDE_data* p = data->parent;
    struct XOMP_mapped_variable* src_element  =  p->new_variables + i;

    copy_mapped_variable(dest_element, src_element);
    offset ++;
  }

  for (i = 0; i < data->parent->inherited_variable_count; i++)
  {
    //copy_mapped_variable(&((data->inherited_variables)[offset]), &( (data->parent->inherited_variables)[i]));
    copy_mapped_variable( (struct XOMP_mapped_variable*) (data->inherited_variables + offset), (struct XOMP_mapped_variable*) (data->parent->inherited_variables + i));
    offset ++;
  }
  assert (offset == data->inherited_variable_count);

}

// Check if an original  variable is already mapped in enclosing data environment, return its device variable's address if yes.
// return NULL if not
void* xomp_deviceDataEnvironmentGetInheritedVariable (void* orig_var, int size)
{
  void * dev_address = NULL; 
  assert (orig_var != NULL);
  int i; 
  // At this point, DDE list should not be empty
  // At least a call to XOMP_Device_Data_Environment_Enter() should have finished before
  assert ( DDE_tail != NULL );
  for (i = 0; i < DDE_tail->inherited_variable_count; i++)
  {
    struct XOMP_mapped_variable* cur_var = DDE_tail->inherited_variables + i; 
    if (cur_var->address == orig_var && cur_var->size == size)
    {
      dev_address = cur_var-> dev_address;
      break;
    }
  } 
  return dev_address; 
}

//! Add a newly mapped variable into the current DDE's new variable list
void xomp_deviceDataEnvironmentAddVariable (void* var_addr, int var_size, void * dev_addr, bool copyTo, bool copyFrom)
{
  // TODO: sanity check to avoid add duplicated variable or inheritable variable
  assert ( DDE_tail != NULL );
  struct XOMP_mapped_variable* mapped_var = DDE_tail->new_variables + DDE_tail->new_variable_count ;
  mapped_var-> address = var_addr; 
  mapped_var-> size = var_size; 
  mapped_var-> dev_address = dev_addr; 
  mapped_var-> copyTo= copyTo; 
  mapped_var-> copyFrom= copyFrom; 
  // now move up the offset
  DDE_tail->new_variable_count ++;
}

// All-in-one function to prepare device variable
void* xomp_deviceDataEnvironmentPrepareVariable(void* original_variable_address, int vsize, bool copy_into, bool copy_back)
{
  void* dev_var_address = NULL; 
  dev_var_address = xomp_deviceDataEnvironmentGetInheritedVariable (original_variable_address, vsize);
  if (dev_var_address == NULL)
  {
    dev_var_address = xomp_deviceMalloc(vsize);
    xomp_deviceDataEnvironmentAddVariable (original_variable_address, vsize, dev_var_address, copy_into, copy_back);

    // The spec says : reuse enclosing data and discard map-type rule.
    // So map-type only matters when no-reuse happens
    if (copy_into)
      xomp_memcpyHostToDevice(dev_var_address, original_variable_address, vsize);
  }
  assert (dev_var_address != NULL);
  return dev_var_address;
}

// Exit current DDE: copy back values if specified, deallocate memory, delete the DDE-data node from the end of the tracking list
void xomp_deviceDataEnvironmentExit()
{
  assert ( DDE_tail != NULL );

  // Deallocate mapped device variables which are allocated by this current DDE
  // Optionally copy the value back to host if specified.
  int i; 
  for (i = 0; i < DDE_tail->new_variable_count; i++)
  {
    struct XOMP_mapped_variable* mapped_var = DDE_tail->new_variables + i;
    void * dev_address = mapped_var->dev_address;
    if (mapped_var->copyFrom)
    {
       xomp_memcpyDeviceToHost(((void *)mapped_var->address),((const void *)mapped_var->dev_address), mapped_var->size);
    }
    // free after copy back!!
    xomp_freeDevice (dev_address); //TODO Will this work without type info? Looks so!
  }

  // Deallocate pre-allocated variable lists
  free (DDE_tail->new_variables);
  free (DDE_tail->inherited_variables);
  
  // Delete the node from the tail
  struct DDE_data * parent = DDE_tail->parent; 
  if (parent != NULL)
  {
    assert (DDE_tail == parent->child); 
    DDE_tail = parent; 
    free (parent->child);
    parent->child = NULL;
  }
  else // last node in the list
  {
    free (DDE_tail);
    DDE_head = NULL;
    DDE_tail = NULL;
  }  
}


